#include "hip/hip_runtime.h"
#include "common.h"
#include "ffi.h"
#include "kernels/kernels.h"
#include "ops.h"

#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <algorithm>
#include <cstddef>
#include <cstdint>

void compute_cov2d_bounds(hipStream_t stream, void **buffers,
                          const char *opaque, std::size_t opaque_len) {
  const Cov2DBoundsDescriptor &d =
      *unpack_descriptor<Cov2DBoundsDescriptor>(opaque, opaque_len);

  const float *__restrict__ covs2d = static_cast<float *>(buffers[0]);

  float *__restrict__ conics = static_cast<float *>(buffers[1]);
  float *__restrict__ radii = static_cast<float *>(buffers[2]);

  constexpr unsigned block_dim = 256;
  const unsigned grid_dim =
      std::min((d.num_points + block_dim - 1) / block_dim, MAX_GRID_DIM);

  kernels::compute_cov2d_bounds<<<grid_dim, block_dim, 0, stream>>>(
      d.num_points, covs2d, conics, radii);

  throw_if_error(hipGetLastError());
}

void project_gaussians_fwd(hipStream_t stream, void **buffers,
                           const char *opaque, std::size_t opaque_len) {

  const ProjectGaussiansFwdDescriptor &d =
      *unpack_descriptor<ProjectGaussiansFwdDescriptor>(opaque, opaque_len);

  const float *__restrict__ means3d = static_cast<float *>(buffers[0]);
  const float *__restrict__ scales = static_cast<float *>(buffers[1]);
  const float *__restrict__ quats = static_cast<float *>(buffers[2]);
  const float *__restrict__ viewmat = static_cast<float *>(buffers[3]);

  float *__restrict__ covs3d_d = static_cast<float *>(buffers[4]);
  float *__restrict__ xys_d = static_cast<float *>(buffers[5]);
  float *__restrict__ depths_d = static_cast<float *>(buffers[6]);
  int *__restrict__ radii_d = static_cast<int *>(buffers[7]);
  float *__restrict__ conics_d = static_cast<float *>(buffers[8]);
  float *__restrict__ compensation_d = static_cast<float *>(buffers[9]);
  std::int32_t *__restrict__ num_tiles_hit_d =
      static_cast<std::int32_t *>(buffers[10]);

  dim3 img_size;
  img_size.x = d.img_shape.first;
  img_size.y = d.img_shape.second;

  dim3 tile_bounds_dim3;
  tile_bounds_dim3.x = int((img_size.x + d.block_width - 1) / d.block_width);
  tile_bounds_dim3.y = int((img_size.y + d.block_width - 1) / d.block_width);
  tile_bounds_dim3.z = 1;

  float4 intrins = {d.f.first, d.f.second, d.c.first, d.c.second};

  constexpr unsigned block_dim = 256;
  const unsigned grid_dim =
      std::min((d.num_points + block_dim - 1) / block_dim, MAX_GRID_DIM);

  kernels::project_gaussians_fwd<<<grid_dim, block_dim, 0, stream>>>(
      d.num_points, (float3 *)means3d, (float3 *)scales, d.glob_scale,
      (float4 *)quats, viewmat, intrins, img_size, tile_bounds_dim3,
      d.block_width, d.clip_thresh,
      // Outputs.
      covs3d_d, (float2 *)xys_d, depths_d, radii_d, (float3 *)conics_d,
      compensation_d, num_tiles_hit_d);
}

void project_gaussians_bwd(hipStream_t stream, void **buffers,
                           const char *opaque, std::size_t opaque_len) {

  const ProjectGaussiansBwdDescriptor &d =
      *unpack_descriptor<ProjectGaussiansBwdDescriptor>(opaque, opaque_len);

  const float *__restrict__ means3d = static_cast<float *>(buffers[0]);
  const float *__restrict__ scales = static_cast<float *>(buffers[1]);
  const float *__restrict__ quats = static_cast<float *>(buffers[2]);
  const float *__restrict__ viewmat = static_cast<float *>(buffers[3]);
  const float *__restrict__ cov3d = static_cast<float *>(buffers[4]);
  const int *__restrict__ radii = static_cast<int *>(buffers[5]);
  const float *__restrict__ conics = static_cast<float *>(buffers[6]);
  const float *__restrict__ compensation = static_cast<float *>(buffers[7]);
  const float *__restrict__ v_xy = static_cast<float *>(buffers[8]);
  const float *__restrict__ v_depth = static_cast<float *>(buffers[9]);
  const float *__restrict__ v_conic = static_cast<float *>(buffers[10]);
  const float *__restrict__ v_compensation = static_cast<float *>(buffers[11]);

  float *__restrict__ v_cov2d = static_cast<float *>(buffers[12]);
  float *__restrict__ v_cov3d = static_cast<float *>(buffers[13]);
  float *__restrict__ v_mean3d = static_cast<float *>(buffers[14]);
  float *__restrict__ v_scale = static_cast<float *>(buffers[15]);
  float *__restrict__ v_quat = static_cast<float *>(buffers[16]);

  dim3 img_size = {d.img_shape.first, d.img_shape.second};
  float4 intrins = {d.f.first, d.f.second, d.c.first, d.c.second};
  const unsigned num_cov3d = d.num_points * 6;

  constexpr unsigned block_dim = 256;
  const unsigned grid_dim =
      std::min((d.num_points + block_dim - 1) / block_dim, MAX_GRID_DIM);

  kernels::project_gaussians_bwd<<<grid_dim, block_dim, 0, stream>>>(
      d.num_points, (float3 *)means3d, (float3 *)scales, d.glob_scale,
      (float4 *)quats, viewmat, intrins, img_size, cov3d, radii,
      (float3 *)conics, compensation, (float2 *)v_xy, v_depth,
      (float3 *)v_conic, v_compensation, (float3 *)v_cov2d, v_cov3d,
      (float3 *)v_mean3d, (float3 *)v_scale, (float4 *)v_quat);
}
